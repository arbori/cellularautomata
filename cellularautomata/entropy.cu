#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"
//#include <stdio.h>

/*!
*
*/
__global__ void BinarySiteEntropyKernel(int* ca, int X, int Y, float* entropyCA) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;

	float states[] = { 0.0, 0.0 };
	float sum = 0.0;

	for (int y = 0; y < Y; y++) {
		sum += 1.0;

		states[ca[x + y*X]] += 1.0;
	}

	states[0] /= sum;
	states[1] /= sum;

	if (states[0] == 0.0f || states[1] == 0.0f) {
		entropyCA[x] = 0.0f;
	}
	else {
		entropyCA[x] = -(states[0] * (log(states[0]) / log(2.0)) + states[1] * (log(states[1]) / log(2.0)));
	}
}

/*!
 *
 */
hipError_t BinarySiteEntropy(int* ca, int X, int Y, float* entropyCA)
{
	int *dev_ca = 0;
	float *dev_entropyCA = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_ca, X * Y * sizeof(int));
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_entropyCA, X * sizeof(float));
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_ca, ca, X * Y * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	BinarySiteEntropyKernel << <1, X >> >(dev_ca, X, Y, dev_entropyCA);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(entropyCA, dev_entropyCA, X * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceReset failed!");
		goto Error;
	}

Error:
	hipFree(dev_ca);
	hipFree(dev_entropyCA);

	return cudaStatus;
}
